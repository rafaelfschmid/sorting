#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

using namespace hipcub;

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

void print(int* host_data, int n) {
	std::cout << "\n";
	for (int i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {

	int num_of_elements;
	int i;

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(int) * num_of_elements;
	int *h_vec = (int *) malloc(mem_size_vec);
	int *h_value = (int *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
		h_value[i] = i;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *d_value, *d_value_out, *d_vec, *d_vec_out;
	void *d_temp = NULL;
	size_t temp_bytes = 0;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value_out, mem_size_vec));

	cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
	cudaTest(
			hipMemcpy(d_value, h_value, mem_size_vec, hipMemcpyHostToDevice));

	hipEventRecord(start);
	hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
			d_value, d_value_out, num_of_elements);
	hipMalloc((void **) &d_temp, temp_bytes);
	hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
			d_value, d_value_out, num_of_elements);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	hipMemcpy(h_value, d_value_out, mem_size_vec, hipMemcpyDeviceToHost);
	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else
		print(h_vec, num_of_elements);

	free(h_vec);
	hipFree(d_vec);
	hipFree(d_vec_out);
	hipFree(d_value);
	hipFree(d_value_out);
	hipFree(d_temp);
	//hipDeviceReset();

	return 0;
}

/***
 * SEGMENTED SORT FUNCIONANDO
 *
 *
 int n = atoi(argv[1]);
 int m = atoi(argv[2]);
 int num_segments = n / m;
 mgpu::standard_context_t context;
 rand_key<int> func(m);

 mgpu::mem_t<int> segs = mgpu::fill_function(func, num_segments, context);
 //mgpu::mem_t<int> segs = mgpu::fill_random(0, n - 1, num_segments, true, context);
 std::vector<int> segs_host = mgpu::from_mem(segs);
 mgpu::mem_t<int> data = mgpu::fill_random(0, pow(2, NUMBER_BITS_SIZE), n,
 false, context);
 mgpu::mem_t<int> values(n, context);
 std::vector<int> data_host = mgpu::from_mem(data);

 //	print(segs_host); print(data_host);

 mgpu::segmented_sort(data.data(), values.data(), n, segs.data(),
 num_segments, mgpu::less_t<int>(), context);

 std::vector<int> sorted = from_mem(data);
 std::vector<int> indices_host = from_mem(values);

 std::cout << "\n";
 //print(segs_host);
 //	print(data_host); print(indices_host);
 *
 */
