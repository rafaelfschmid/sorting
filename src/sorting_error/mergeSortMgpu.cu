#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <moderngpu/kernel_mergesort.hxx>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
//#include <cstdlib>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

using namespace std::chrono;

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

void print(int* host_data, int n) {
	std::cout << "\n";
	for (int i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {

	int num_of_elements;
	int i;

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(int) * num_of_elements;
	int *h_vec = (int *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec[i]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *d_vec;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));

	hipEventRecord(start);
	mgpu::standard_context_t context;
	mgpu::mergesort(d_vec, num_of_elements, mgpu::less_t<int>(), context);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	cudaTest(hipMemcpy(h_vec, d_vec, mem_size_vec, hipMemcpyDeviceToHost));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else
		print(h_vec, num_of_elements);

	/*
	 * NUNCA usar hipDeviceReset nesse código
	 * hipDeviceReset();
	 * */

	free(h_vec);
	hipFree(d_vec);

	return 0;
}

/***
 * SEGMENTED SORT FUNCIONANDO
 *
 *
 int n = atoi(argv[1]);
 int m = atoi(argv[2]);
 int num_segments = n / m;
 mgpu::standard_context_t context;
 rand_key<int> func(m);

 mgpu::mem_t<int> segs = mgpu::fill_function(func, num_segments, context);
 //mgpu::mem_t<int> segs = mgpu::fill_random(0, n - 1, num_segments, true, context);
 std::vector<int> segs_host = mgpu::from_mem(segs);
 mgpu::mem_t<int> data = mgpu::fill_random(0, pow(2, NUMBER_BITS_SIZE), n,
 false, context);
 mgpu::mem_t<int> values(n, context);
 std::vector<int> data_host = mgpu::from_mem(data);

 //	print(segs_host); print(data_host);

 mgpu::segmented_sort(data.data(), values.data(), n, segs.data(),
 num_segments, mgpu::less_t<int>(), context);

 std::vector<int> sorted = from_mem(data);
 std::vector<int> indices_host = from_mem(values);

 std::cout << "\n";
 //print(segs_host);
 //	print(data_host); print(indices_host);
 *
 */
